#include "hip/hip_runtime.h"
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd,int width)
{
  // Calculate the row index of the Pd element and M
  int Row = blockIdx.y*TILE_WIDTH + threadIdx.y;
  // Calculate the column index of Pd and N
  int Col = blockIdx.x*TILE_WIDTH + threadIdx.x;
  float Pvalue = 0;
  // each thread computes one element of the block sub-matrix
  for (int k = 0; k < width; ++k) {
    Pvalue += Md[Row*width+k] * Nd[k*width+Col];
  }
  Pd[Row*width+Col] = Pvalue;
}

// Matrix multiplication kernel
// -- per thread code
__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd,int width)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Pvalue is used to store the
  // element of the matrix
  // that is computed by the thread
  float Pvalue = 0;
  for (int k = 0; k < width; ++k) {
    float Melement = Md[ty*width+k];
    float Nelement = Nd[k*width+tx];
    Pvalue += Melement * Nelement;
  }
  Pd[ty*width+tx] = Pvalue;
}

// Setup the execution configuration
dim3 dimGrid(width/TILE_WIDTH, width/TILE_WIDTH);
dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
// Launch the device computation threads!
MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

void MatrixMultiplication(float* M, float* N, float* P, int width)
{
  int size = width*width*sizeof(float);
  float* Md, Nd, Pd;
  // 1. Allocate and Load M, N to device memory
  hipMalloc((void**)&Md, size);
  hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
  hipMalloc((void**)&Nd, size);
  hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);
  // Allocate P on the device
  hipMalloc((void**)&Pd, size);
  // 2. Kernel invocation code -- to be shown later
  // 3. Read P from the device
  hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);
  // Free device matrices
  hipFree(Md); hipFree(Nd); hipFree (Pd);
}
