/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>
#include <time.h>
#include "support.h"
#include "kernel.cu"

int main(int argc, char**argv) {

    Timer timer;
    hipError_t cuda_ret;
    time_t t;


    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    unsigned int n;
    if(argc == 1) {
        n = 10000;
    } else if(argc == 2) {
        n = atoi(argv[1]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./vecadd               # Vector of size 10,000 is used"
           "\n    Usage: ./vecadd <m>           # Vector of size m is used"
           "\n");
        exit(0);
    }

    /* Intializes random number generator */
    srand((unsigned) time(&t));


    float* A_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { A_h[i] = (rand()%100)/100.00; }

    float* B_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { B_h[i] = (rand()%100)/100.00; }

    float* C_h = (float*) malloc( sizeof(float)*n );

    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("    Vector size = %u\n", n);

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);

    // INSERT CODE HERE
    // Allocate and Load A_h, B_h, and C_h to device memory
    hipMalloc((void**)&A_h, sizeof(float)*n);
    hipMalloc((void**)&B_h, sizeof(float)*n);
    hipMalloc((void**)&C_h, sizeof(float)*n);



    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    hipMemcpy(A_h, A_d, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(B_h, B_d, sizeof(float)*n, hipMemcpyHostToDevice);



    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------

    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
    dim3 dimBlock(512);
    dim3 dimGrid(n/512);

    vecAddKernel<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, n);


    cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) {
		printf("Unable to launch kernel\n");
		exit(-1);
    }
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE

    hipMemcpy(C_h, C_d, sizeof(float)*n, hipMemcpyDeviceToHost);


    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Verify correctness -----------------------------------------------------

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, n);

    // Free memory ------------------------------------------------------------

    free(A_h);
    free(B_h);
    free(C_h);

    //INSERT CODE HERE

    hipFree(A_h); hipFree(B_h); hipFree (C_h);



    return 0;

}
