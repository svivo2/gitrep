#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

__global__ void vecAddKernel(float* A_d, float* B_d, float* C_d, int n) {

    // Calculate global thread index based on the block and thread indices ----

    //INSERT KERNEL CODE HERE

    int tx = threadIdx.x;


    // Use global index to determine which elements to read, add, and write ---

    //INSERT KERNEL CODE HERE

    C_d[n*tx] = A_d[tx*n] + B_H[tx*n]


}
