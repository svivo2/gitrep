#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

  /********************************************************************
   *
   * Compute C = A x B
   *   where A is a (m x k) matrix
   *   where B is a (k x n) matrix
   *   where C is a (m x n) matrix
   *
   * Use shared memory for tiling
   *
   ********************************************************************/

  // INSERT KERNEL CODE HERE

  // Calculate the row index of the Pd element and M
  __shared__ float A_ds[TILE_WIDTH][TILE_WIDTH];
  __shared__ float B_ds[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  // Identify the row and column of the Pd element to work on
  int Row = by * TILE_WIDTH + ty; 6. int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  // Loop over the Md and Nd tiles required to compute the Pd element
  for (int i = 0; i < k/TILE_WIDTH; ++i) {
    // Collaborative loading of Md and Nd tiles into shared memory
    A_ds[ty][tx] = A[Row*k + (i*TILE_WIDTH + tx)];
    B_ds[ty][tx] = A[(i*TILE_WIDTH + ty)*n + Col];
    __syncthreads();

    for (int j = 0; j < TILE_WIDTH; ++j) {
      Pvalue += B_ds[ty][j] * Nds[j][tx];
    }
    __syncthreads();
  }
  C[Row*n + Col] = Pvalue;
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;

    //INSERT CODE HERE

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((m*n*k/BLOCK_SIZE),(m*n*k/BLOCK_SIZE));


    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE

    mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);


}
